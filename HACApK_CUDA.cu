#include "hip/hip_runtime.h"
// -*- c++ -*-
#include	<stdio.h>
#include	<stdlib.h>
#include	<time.h>
#include	"omp.h"
#include	"mpi.h"
#include	"HACApK_MAGMA.h"
//#include        "magma_dlapack.h"

__global__ void cuda_matvec_a1
(int kt, int ndt, int nstrtt, double *d_zbut, double *d_a1, double *d_zu)
{
  int il, it, itt, itl;
  for(il=0; il<kt; il++){
    for(it=0; it<ndt; it++){
      itt=it+nstrtt-1;
      itl=it+il*ndt; 
      d_zbut[il] += d_a1[itl]*d_zu[itt];
    }
  }
}
template <int THREADS_PER_BLOCK>
__global__ void cuda_matvec_a1_2
(int kt, int ndt, int nstrtt, double *d_zbut, double *d_a1, double *d_zu)
{
  int il, it, itt, itl;
  int gid = blockIdx.x;
  int glen = gridDim.x;
  int tid = threadIdx.x;
  int tlen = blockDim.x;
  double tmp=0.0;
  __shared__ double smTmp[THREADS_PER_BLOCK*2];
  for(il=gid; il<kt; il+=glen){
    tmp = 0.0;
    for(it=tid; it<ndt; it+=tlen){
      itt=it+nstrtt-1;
      itl=it+il*ndt; 
      tmp += d_a1[itl]*d_zu[itt];
    }
    smTmp[tid] = tmp;
    smTmp[tid+tlen] = 0.0;
    __syncthreads();
    if(THREADS_PER_BLOCK > 512){    if(tid<512)smTmp[tid] = tmp = tmp + smTmp[tid+512];    __syncthreads();  }
    if(THREADS_PER_BLOCK > 256){    if(tid<256)smTmp[tid] = tmp = tmp + smTmp[tid+256];    __syncthreads();  }
    if(THREADS_PER_BLOCK > 128){    if(tid<128)smTmp[tid] = tmp = tmp + smTmp[tid+128];    __syncthreads();  }
    if(THREADS_PER_BLOCK >  64){    if(tid< 64)smTmp[tid] = tmp = tmp + smTmp[tid+ 64];    __syncthreads();  }
    if(THREADS_PER_BLOCK >  32){    if(tid< 32)smTmp[tid] = tmp = tmp + smTmp[tid+ 32];    __syncthreads();  }
    if(tid<32){
      for (int offset = warpSize/2; offset > 0; offset /= 2){
	tmp += __shfl_down(tmp, offset);
      }
    }
    if(tid==0)d_zbut[il] += tmp;
  }
}
__global__ void cuda_matvec_a2
(int kt, int ndl, int nstrtl, double *d_zaut, double *d_a2tmp, double *d_zbut)
{
  int il, it, ill, itl;
  for(il=0; il<kt; il++){
    for(it=0; it<ndl; it++){
      ill=it+nstrtl-1;
      itl=it+il*ndl; 
      d_zaut[ill] += d_a2tmp[itl]*d_zbut[il];
    }
  }
}
__global__ void cuda_matvec_a2_2a
(int kt, int ndl, int nstrtl, double *d_zaut, double *d_a2tmp, double *d_zbut)
{
  int il, it, ill, itl;
  int gid = blockIdx.x;
  int glen = gridDim.x;
  int tid = threadIdx.x;
  int tlen = blockDim.x;
  for(il=gid; il<kt; il+=glen){
    for(it=tid; it<ndl; it+=tlen){
      ill=it+nstrtl-1;
      itl=it+il*ndl;
      atomicAdd(&d_zaut[ill], d_a2tmp[itl]*d_zbut[il]);
    }
  }
}
template <int THREADS_PER_BLOCK>
__global__ void cuda_matvec_a2_2b
(int kt, int ndl, int nstrtl, double *d_zaut, double *d_a2tmp, double *d_zbut)
{
  int il, it, ill, itl;
  int gid = blockIdx.x;
  int glen = gridDim.x;
  int tid = threadIdx.x;
  int tlen = blockDim.x;
  double tmp=0.0;
  __shared__ double smTmp[THREADS_PER_BLOCK*2];
  for(it=gid; it<ndl; it+=glen){
    tmp = 0.0;
    ill=it+nstrtl-1;
    for(il=tid; il<kt; il+=tlen){
      itl=it+il*ndl; 
      tmp += d_a2tmp[itl]*d_zbut[il];
    }
    smTmp[tid] = tmp;
    smTmp[tid+tlen] = 0.0;
    __syncthreads();
    if(THREADS_PER_BLOCK > 512){    if(tid<512)smTmp[tid] = tmp = tmp + smTmp[tid+512];    __syncthreads();  }
    if(THREADS_PER_BLOCK > 256){    if(tid<256)smTmp[tid] = tmp = tmp + smTmp[tid+256];    __syncthreads();  }
    if(THREADS_PER_BLOCK > 128){    if(tid<128)smTmp[tid] = tmp = tmp + smTmp[tid+128];    __syncthreads();  }
    if(THREADS_PER_BLOCK >  64){    if(tid< 64)smTmp[tid] = tmp = tmp + smTmp[tid+ 64];    __syncthreads();  }
    if(THREADS_PER_BLOCK >  32){    if(tid< 32)smTmp[tid] = tmp = tmp + smTmp[tid+ 32];    __syncthreads();  }
    if(tid<32){
      for (int offset = warpSize/2; offset > 0; offset /= 2){
	tmp += __shfl_down(tmp, offset);
      }
    }
    if(tid==0)d_zaut[ill] += tmp;
  }
}

__global__ void cuda_matvec_s
(int ndl, int ndt, int nstrtl, int nstrtt, double *d_zaut, double *d_a1, double *d_zu)
{
  int il, it, ill, itt, itl;
  for(il=0; il<ndl; il++){
    ill=il+nstrtl-1; 
    for(it=0; it<ndt; it++){
      itt=it+nstrtt-1; 
      itl=it+il*ndt;
      d_zaut[ill] += d_a1[itl]*d_zu[itt];
    }
  }
}
template <int THREADS_PER_BLOCK>
__global__ void cuda_matvec_s_2
(int ndl, int ndt, int nstrtl, int nstrtt, double *d_zaut, double *d_a1, double *d_zu)
{
  int il, it, ill, itt, itl;
  int gid = blockIdx.x;
  int glen = gridDim.x;
  int tid = threadIdx.x;
  int tlen = blockDim.x;
  double tmp=0.0;
  __shared__ double smTmp[THREADS_PER_BLOCK*2];
  for(il=gid; il<ndl; il+=glen){
    tmp = 0.0;
    ill=il+nstrtl-1;
    for(it=tid; it<ndt; it+=tlen){
      itt=it+nstrtt-1; 
      itl=it+il*ndt;
      tmp += d_a1[itl]*d_zu[itt];
    }
    smTmp[tid] = tmp;
    smTmp[tid+tlen] = 0.0;
    __syncthreads();
    if(THREADS_PER_BLOCK > 512){    if(tid<512)smTmp[tid] = tmp = tmp + smTmp[tid+512];    __syncthreads();  }
    if(THREADS_PER_BLOCK > 256){    if(tid<256)smTmp[tid] = tmp = tmp + smTmp[tid+256];    __syncthreads();  }
    if(THREADS_PER_BLOCK > 128){    if(tid<128)smTmp[tid] = tmp = tmp + smTmp[tid+128];    __syncthreads();  }
    if(THREADS_PER_BLOCK >  64){    if(tid< 64)smTmp[tid] = tmp = tmp + smTmp[tid+ 64];    __syncthreads();  }
    if(THREADS_PER_BLOCK >  32){    if(tid< 32)smTmp[tid] = tmp = tmp + smTmp[tid+ 32];    __syncthreads();  }
    if(tid<32){
      for (int offset = warpSize/2; offset > 0; offset /= 2){
	tmp += __shfl_down(tmp, offset);
      }
    }
    if(tid==0)d_zaut[ill] += tmp;
  }
}

void  c_hacapk_adot_body_lfmtx_cuda_calc
(double *zau, stc_HACApK_leafmtxp *st_leafmtxp, double *zu, double *zbu,
 double *time_batch, double *time_set, double *time_copy, int nd) {
  register int ip,il,it;
  int nlf,ndl,ndt,nstrtl,nstrtt,kt,itl,itt,ill;
  int st_lf_stride = st_leafmtxp->st_lf_stride;
  size_t a1size;
  int ith, nths, nthe;
  double *zaut, *zbut;
  int ls, le;
  int i;


  double *d_zaut, *d_zbut;
  double *d_zau, *d_zu;
  double *d_a1, *d_a2tmp;
  hipMalloc(&d_zaut, sizeof(double)*nd);
  hipMalloc(&d_zbut, sizeof(double)*st_leafmtxp->ktmax);
  hipMalloc(&d_zau, sizeof(double)*nd);
  hipMalloc(&d_zu, sizeof(double)*nd);

  for(i=0;i<nd;i++)zau[i]=0.0;
  hipMemcpy(d_zau, zau, sizeof(double)*nd, hipMemcpyHostToDevice);
  hipMemcpy(d_zu, zu, sizeof(double)*nd, hipMemcpyHostToDevice);

  nlf=st_leafmtxp->nlf;
  //fprintf(stderr,"nlf=%d \n",nlf);

  zaut = (double*)malloc(sizeof(double)*nd);
  for(il=0;il<nd;il++)zaut[il]=0.0;
  hipMemcpy(d_zaut, zaut, sizeof(double)*nd, hipMemcpyHostToDevice);
  //printf("st_leafmtxp->ktmax = %d\n",st_leafmtxp->ktmax);
  zbut = (double*)malloc(sizeof(double)*st_leafmtxp->ktmax);
  ls = nd;
  le = 1;
  for(ip=0; ip<nlf; ip++){
    //ip=0;{
    /**/
    stc_HACApK_leafmtx *sttmp;
    sttmp = (stc_HACApK_leafmtx *)((size_t)((void *)(st_leafmtxp->st_lf)) + st_lf_stride * ip);
    //fprintf(stderr, "%d: %p\n", ip, sttmp);
    /**/

    ndl   =sttmp->ndl; 
    ndt   =sttmp->ndt;
    nstrtl=sttmp->nstrtl; 
    nstrtt=sttmp->nstrtt;
    //fprintf(stderr,"ip=%d, ndl=%d, ndt=%d, nstrtl=%d, nstrtt=%d \n",ip,ndl,ndt,nstrtl,nstrtt);
    hipMalloc(&d_a2tmp, sizeof(double)*ndl*ndt);
    if(nstrtl<ls)ls=nstrtl;
    if(nstrtl+ndl-1>le)le=nstrtl+ndl-1;
    //printf("DBG: ltmtx=%d\n",sttmp->ltmtx);
    if(sttmp->ltmtx==1){
      /**/
      double *a2tmp = (double *)((size_t)((void*)(sttmp->a1))+sttmp->a1size);
      /**/
      kt=sttmp->kt;
      for(il=0;il<kt;il++)zbut[il]=0.0;
      hipMemcpy(d_zbut, zbut, sizeof(double)*kt, hipMemcpyHostToDevice);
      hipMemcpy(d_a2tmp, a2tmp, sizeof(double)*ndl*ndt, hipMemcpyHostToDevice);
      hipMalloc(&d_a1, sizeof(double)*ndt*kt);
      hipMemcpy(d_a1, sttmp->a1, sizeof(double)*ndt*kt, hipMemcpyHostToDevice);
      //cuda_matvec_a1<<<1,1>>>(kt,ndt,nstrtt,d_zbut,d_a1,d_zu);
      cuda_matvec_a1_2<128><<<112,128>>>(kt,ndt,nstrtt,d_zbut,d_a1,d_zu);
      /*
	for(il=0; il<kt; il++){
	  for(it=0; it<ndt; it++){
	    itt=it+nstrtt-1;
	    itl=it+il*ndt; 
	    zbut[il] += sttmp->a1[itl]*zu[itt];
	  }
	}
      */
      //cuda_matvec_a2<<<1,1>>>(kt,ndl,nstrtl,d_zaut,d_a2tmp,d_zbut);
      //cuda_matvec_a2_2a<<<112,128>>>(kt,ndl,nstrtl,d_zaut,d_a2tmp,d_zbut);
      cuda_matvec_a2_2b<128><<<112,128>>>(kt,ndl,nstrtl,d_zaut,d_a2tmp,d_zbut);
      /*
	for(il=0; il<kt; il++){
	  for(it=0; it<ndl; it++){
	    ill=it+nstrtl-1;
	    itl=it+il*ndl; 
	    zaut[ill] += a2tmp[itl]*zbut[il];
	  }
	}
      */
    } else if(sttmp->ltmtx==2){
      hipMalloc(&d_a1, sizeof(double)*ndt*ndl);
      hipMemcpy(d_a1, sttmp->a1, sizeof(double)*ndt*ndl, hipMemcpyHostToDevice);
      //cuda_matvec_s<<<1,1>>>(ndl,ndt,nstrtl,nstrtt,d_zaut,d_a1,d_zu);
      cuda_matvec_s_2<128><<<112,128>>>(ndl,ndt,nstrtl,nstrtt,d_zaut,d_a1,d_zu);
      /*
	for(il=0; il<ndl; il++){
	  ill=il+nstrtl-1; 
	  for(it=0; it<ndt; it++){
	    itt=it+nstrtt-1; 
	    itl=it+il*ndt;
	    zaut[ill] += sttmp->a1[itl]*zu[itt];
	  }
	}
      */
    }
    hipFree(d_a1);
    hipFree(d_a2tmp);
  }
  hipMemcpy(zaut, d_zaut, sizeof(double)*nd, hipMemcpyDeviceToHost);
  for(il=ls-1;il<=le-1;il++){
    zau[il] += zaut[il];
  }
  /*
    for(il=ls-1;il<=le-1;il++){
    #pragma omp atomic
    zau[il] += zaut[il];
    }
  */
  free(zaut); free(zbut);
  hipFree(d_zaut); hipFree(d_zbut); hipFree(d_zau); hipFree(d_zu);
}

void c_hacapk_adot_cax_lfmtx_cuda_comm
(double *zau, stc_HACApK_lcontrol *st_ctl,
 double *wws, double *wwr, int *isct, int *irct, int nd, double *time_mpi) {
  int ione = 1;
  double one = 1.0;

  double tic;
  int *lpmd = (int*)((size_t)((void*)st_ctl->param) + st_ctl->lpmd_offset); 
  int mpinr = lpmd[2]; 
  int nrank = lpmd[1]; 
  int i;
   
  if (nrank > 1) {
    int *lsp = (int*)((size_t)((void*)st_ctl->param) + st_ctl->lsp_offset);
    int *lnp = (int*)((size_t)((void*)st_ctl->param) + st_ctl->lnp_offset);
    MPI_Comm icomm = MPI_COMM_WORLD;

    int ic;
    int ncdp = (mpinr+1)%nrank;       // my destination neighbor
    int ncsp = (mpinr+nrank-1)%nrank; // my source neighbor
    isct[0] = lnp[mpinr];
    isct[1] = lsp[mpinr];

    // copy local vector to send buffer
    //dlacpy_( "F", &lnp[mpinr], &ione, &zau[lsp[mpinr]-1], &lnp[mpinr], wws, &lnp[mpinr] );
    for(i=0;i<lnp[mpinr];i++)wws[i]=zau[lsp[mpinr]-1+i];
    for (ic=1; ic<nrank; ic++) {
      MPI_Status stat;
      tic = MPI_Wtime();
      // read offset/size from structure
      int nctp = (ncsp-ic+nrank+1)%nrank; // where it came from
      irct[0] = lnp[nctp];
      irct[1] = lsp[nctp];
      
      MPI_Status stats[2];
      MPI_Request reqs[2];
      if (MPI_SUCCESS != MPI_Isend(wws, isct[0], MPI_DOUBLE, ncdp, nrank+ic, MPI_COMM_WORLD, &reqs[0])) 
	printf( "MPI_Isend failed\n" );
      if (MPI_SUCCESS != MPI_Irecv(wwr, irct[0], MPI_DOUBLE, ncsp, nrank+ic, MPI_COMM_WORLD, &reqs[1]))
	printf( "MPI_Irecv failed\n" );
      if (MPI_SUCCESS != MPI_Waitall(2, reqs, stats))
	printf( "MPI_Waitall failed\n" );
      
      *time_mpi += (MPI_Wtime()-tic);
      //blasf77_daxpy( &irct[0], &one, wwr, &ione, &zau[irct[1]-1], &ione );
      for(i=0;i<irct[0];i++)zau[irct[1]-1+i]+=wwr[i];

      //dlacpy_( "F", &irct[0], &ione, wwr, &irct[0], wws, &irct[0] );
      for(i=0;i<irct[0];i++)wws[i]=wwr[i];
      isct[0] = irct[0];
      isct[1] = irct[1];
    }
  }
}

extern "C"
void c_hacapk_bicgstab_cax_lfmtx_cuda_
(stc_HACApK_leafmtxp *st_leafmtxp, stc_HACApK_lcontrol *st_ctl,
 double *u, double *b, double*param, int *nd, int *nstp, int *lrtrn) {
  // local constants
  int ione = 1;
  double zero =  0.0;
  double one  =  1.0;
  double mone = -1.0;
  // local arrays
  double *zr, *zshdw, *zp, *zt, *zkp, *zakp, *zkt, *zakt;
  double *wws, *wwr;
  int *lpmd = (int*)((size_t)((void*)st_ctl->param) + st_ctl->lpmd_offset);
  int isct[2], irct[2];
  // local variables
  double eps, alpha, beta, zeta, zz, zden, znorm, znormold, bnorm, zrnorm;
  double en_measure_time, st_measure_time, time;
  int info, step, mstep;
  int mpinr, nrank, ierr;
  double time_spmv, time_mpi, time_batch, time_set, time_copy, tic;
  int i, tid;
  MPI_Comm icomm = MPI_COMM_WORLD; //lpmd[0];
  mstep = param[82];
  eps = param[90];
  mpinr = lpmd[2];
  nrank = lpmd[1];
  MPI_Barrier( icomm );

  wws = (double*)malloc((*nd) * sizeof(double));
  wwr = (double*)malloc((*nd) * sizeof(double));

  zt = (double*)malloc((*nd) * sizeof(double));
  zr = (double*)malloc((*nd) * sizeof(double));
  zp = (double*)malloc((*nd) * sizeof(double));
  zkp = (double*)malloc((*nd) * sizeof(double));
  zakp = (double*)malloc((*nd) * sizeof(double));
  zkt = (double*)malloc((*nd) * sizeof(double));
  zakt= (double*)malloc((*nd) * sizeof(double));
  zshdw = (double*)malloc((*nd) * sizeof(double));
  // copy matrix to GPU
  //c_hacapk_adot_body_lfcpy_batch_sorted_(nd, st_leafmtxp);

  time_spmv = 0.0;
  time_mpi = 0.0;
  time_batch = 0.0;
  time_set = 0.0;
  time_copy = 0.0;
  MPI_Barrier( icomm );
  st_measure_time = MPI_Wtime();
  // init
  alpha = 0.0; beta = 0.0; zeta = 0.0;
  zz = 0.0;
#pragma omp parallel for reduction(+:zz)
  for(i=0;i<(*nd);i++){zz += b[i]*b[i];}
  bnorm=sqrt(zz);
  printf("bnorm:%e\n",bnorm);
#pragma omp parallel for
  for(i=0;i<(*nd);i++)zr[i]=b[i];
  //  .. MATVEC ..
  tic = MPI_Wtime();
  //for(i=0;i<(*nd);i++)zshdw[i]=0.0;
  c_hacapk_adot_body_lfmtx_cuda_calc(zshdw,st_leafmtxp,u,wws, &time_batch,&time_set,&time_copy,*nd);
  /*
  {
    FILE *F;
    F=fopen("cuda1.dat","w");
    for(i=0;i<(*nd);i++){
      fprintf(F,"%e\n",zshdw[i]);
    }
  }
  */
  time_spmv += (MPI_Wtime()-tic);
  c_hacapk_adot_cax_lfmtx_cuda_comm(zshdw, st_ctl, wws, wwr, isct, irct, *nd, &time_mpi);
  //
  /*
  {
    FILE *F;
    F=fopen("cuda2.dat","w");
    for(i=0;i<(*nd);i++){
      fprintf(F,"%e\n",zshdw[i]);
    }
  }
  */
#pragma omp parallel for
  for(i=0;i<(*nd);i++){
    zr[i]+=mone*zshdw[i];
    zshdw[i]=zr[i];
  }
  zrnorm = 0.0;
#pragma omp parallel for reduction(+:zrnorm)
  for(i=0;i<(*nd);i++)zrnorm += zr[i]*zr[i];
  zrnorm = sqrt(zrnorm);
  printf("zrnorm:%e",zrnorm);
  //return;
  if (mpinr == 0) {
	printf( "\n ** BICG (c version, CUDA) **\n" );
	printf( "\nOriginal relative residual norm = %.2e/%.2e = %.2e\n",zrnorm,bnorm,zrnorm/bnorm );
	printf( "c_HACApK_bicgstab_cax_lfmtx_cuda start\n" );
  }
  for ( step=1; step<=mstep; step++ ) {
    //for(step=1; step<=1; step++){
	if (zrnorm/bnorm < eps) break;
	// zp(:nd) = zr(:nd) + beta*(zp(:nd) - zeta*zakp(:nd))
	if (beta == zero) {
#pragma omp parallel for
	  for(i=0;i<(*nd);i++)zp[i]=zr[i];
	} else {
#pragma omp parallel for
	  for(i=0;i<(*nd);i++){
	    zp[i] = zr[i] + beta * (zp[i] + zeta*zakp[i]);
	  }
	}
	/*
	{
	  FILE *F;
	  F=fopen("cuda-zp.dat","w");
	  for(i=0;i<(*nd);i++)fprintf(F,"%e\n", zp[i]);
	  fclose(F);
	}
	*/
	// zkp(:nd) = zp(:nd)
#pragma omp parallel for
	for(i=0;i<(*nd);i++)zkp[i]=zp[i];
	//  .. MATVEC ..
	//for(i=0;i<(*nd);i++)zakp[i]=0.0;
	tic = MPI_Wtime();
	c_hacapk_adot_body_lfmtx_cuda_calc(zakp,st_leafmtxp,zkp,wws, &time_batch,&time_set,&time_copy,*nd);
	time_spmv += (MPI_Wtime()-tic);
	/*
	{
	  FILE *F;
	  F=fopen("cuda-zakp.dat","w");
	  for(i=0;i<(*nd);i++)fprintf(F,"%e\n", zakp[i]);
	  fclose(F);
	}
	*/
	c_hacapk_adot_cax_lfmtx_cuda_comm(zakp,st_ctl,wws,wwr,isct,irct,*nd, &time_mpi);
	//
	znorm = 0.0;
#pragma omp parallel for reduction(+:znorm)
	for(i=0;i<(*nd);i++)znorm += zshdw[i]*zr[i];
	zden = 0.0;
#pragma omp parallel for reduction(+:zden)
	for(i=0;i<(*nd);i++)zden += zshdw[i]*zakp[i];
	/*
	{
	  FILE *F;
	  F=fopen("cuda.dat","w");
	  fprintf(F,"%e %e\n",znorm, zden);
	  fclose(F);
	}
	*/
	alpha = -znorm/zden;
	znormold = znorm;
	// zt(:nd) = zr(:nd) - alpha*zakp(:nd)
#pragma omp parallel for
	for(i=0;i<(*nd);i++)zt[i]=zr[i]+alpha*zakp[i];
	alpha = -alpha;
	// zkt(:nd) = zt(:nd)
#pragma omp parallel for
	for(i=0;i<(*nd);i++)zkt[i]=zt[i];
	//  .. MATVEC ..
	//for(i=0;i<(*nd);i++)zakt[i]=0.0;
	tic = MPI_Wtime();
	c_hacapk_adot_body_lfmtx_cuda_calc(zakt,st_leafmtxp,zkt,wws, &time_batch,&time_set,&time_copy,*nd);
	time_spmv += (MPI_Wtime()-tic);
	c_hacapk_adot_cax_lfmtx_cuda_comm(zakt,st_ctl,wws,wwr,isct,irct,*nd, &time_mpi);
	//
	znorm = 0.0;
#pragma omp parallel for reduction(+:znorm)
	for(i=0;i<(*nd);i++)znorm += zakt[i]*zt[i];
	zden = 0.0;
#pragma omp parallel for reduction(+:zden)
	for(i=0;i<(*nd);i++)zden += zakt[i]*zakt[i];
	zeta = znorm/zden;
	/*
	{
	  FILE *F;
	  F=fopen("cuda.dat","a");
	  fprintf(F,"%e %e\n",znorm, zden);
	  fclose(F);
	}
	*/
	// u(:nd) = u(:nd) + alpha*zkp(:nd) + zeta*zkt(:nd)
#pragma omp parallel for
	for(i=0;i<(*nd);i++){
	  u[i] += alpha*zkp[i] + zeta*zkt[i];
	}
	// zr(:nd) = zt(:nd) - zeta*zakt(:nd)
	zeta = -zeta;
#pragma omp parallel
	for(i=0;i<(*nd);i++){
	  zr[i]=zt[i] + zeta*zakt[i];
	}
	// beta = alpha/zeta * HACApK_dotp_d(nd,zshdw,zr)/znormold;
	beta = 0.0;
#pragma omp parallel for reduction(+:beta)
	for(i=0;i<(*nd);i++)beta += zshdw[i]*zr[i];
	beta = -alpha/zeta * beta/znormold;
	zrnorm = 0.0;
#pragma omp parallel for reduction(+:zrnorm)
	for(i=0;i<(*nd);i++)zrnorm += zr[i]*zr[i];
        zrnorm = sqrt(zrnorm);
        *nstp = step;
        en_measure_time = MPI_Wtime();
        time = en_measure_time - st_measure_time;
        if (st_ctl->param[0] > 0 && mpinr == 0) {
            printf( " %d: time=%.2e log10(zrnorm/bnorm)=log10(%.2e/%.2e)=%.2e\n",step,time,zrnorm,bnorm,log10(zrnorm/bnorm) );
        }
    }
    MPI_Barrier( icomm );
    en_measure_time = MPI_Wtime();
    time = en_measure_time - st_measure_time;
    if (st_ctl->param[0] > 0) {
        //printf( " End: %d, %.2e\n",mpinr,time );
	  for(i=0;i<nrank;i++){
		if(i==mpinr){
			printf( "C-CUDA  %d  BiCG        = %.5e\n", i, time );
			printf( "C-CUDA  %d  time_mpi   = %.5e\n", i, time_mpi );
			printf( "C-CUDA  %d  time_matvec  = %.5e\n", i, time_spmv );
			printf( "C-CUDA  %d  >time_copy  = %.5e\n", i, time_copy );
			printf( "C-CUDA  %d  >time_set   = %.5e\n", i, time_set );
			printf( "C-CUDA  %d  >time_batch = %.5e\n", i, time_batch );
		}
		MPI_Barrier( icomm );
	  }
    }
    // delete matrix
    //c_hacapk_adot_body_lfdel_batch_(st_leafmtxp);

    // free cpu memory
    free(wws);
    free(wwr);

    free(zt);
    free(zr);
    free(zp);
    free(zkp);
    free(zakp);
    free(zkt);
    free(zakt);
    free(zshdw);
}


void  c_hacapk_adot_body_lfmtx_warp_calc
(double *zau, stc_HACApK_leafmtxp *st_leafmtxp, double *zu, double *zbu,
 double *time_batch, double *time_set, double *time_copy, int nd) {
  register int ip,il,it;
  int nlf,ndl,ndt,nstrtl,nstrtt,kt,itl,itt,ill;
  int st_lf_stride = st_leafmtxp->st_lf_stride;
  size_t a1size;
  int ith, nths, nthe;
  double *zaut, *zbut;
  int ls, le;
  int i;

#pragma omp for
  for(i=0;i<nd;i++)zau[i]=0.0;

  nlf=st_leafmtxp->nlf;
  //fprintf(stderr,"nlf=%d \n",nlf);

  zaut = (double*)malloc(sizeof(double)*nd);
  for(il=0;il<nd;il++)zaut[il]=0.0;
  //printf("st_leafmtxp->ktmax = %d\n",st_leafmtxp->ktmax);
  zbut = (double*)malloc(sizeof(double)*st_leafmtxp->ktmax);
  ls = nd;
  le = 1;
  for(ip=0; ip<nlf; ip++){
	/**/
	stc_HACApK_leafmtx *sttmp;
	sttmp = (stc_HACApK_leafmtx *)((size_t)((void *)(st_leafmtxp->st_lf)) + st_lf_stride * ip);
	//fprintf(stderr, "%d: %p\n", ip, sttmp);
	/**/

	ndl   =sttmp->ndl; 
	ndt   =sttmp->ndt;
	nstrtl=sttmp->nstrtl; 
	nstrtt=sttmp->nstrtt;
	//fprintf(stderr,"ip=%d, ndl=%d, ndt=%d, nstrtl=%d, nstrtt=%d \n",ip,ndl,ndt,nstrtl,nstrtt);
	if(nstrtl<ls)ls=nstrtl;
	if(nstrtl+ndl-1>le)le=nstrtl+ndl-1;
	if(sttmp->ltmtx==1){
	  /**/
	  double *a2tmp = (double *)((size_t)((void*)(sttmp->a1))+sttmp->a1size);
	  /**/
	  kt=sttmp->kt;
	  for(il=0;il<kt;il++)zbut[il]=0.0;
	  for(il=0; il<kt; il++){
		for(it=0; it<ndt; it++){
		  itt=it+nstrtt-1;
		  itl=it+il*ndt; 
		  zbut[il] += sttmp->a1[itl]*zu[itt];
		}
	  }
	  for(il=0; il<kt; il++){
		for(it=0; it<ndl; it++){
		  ill=it+nstrtl-1;
		  itl=it+il*ndl; 
		  zaut[ill] += a2tmp[itl]*zbut[il];
		}
	  }
	} else if(sttmp->ltmtx==2){
	  for(il=0; il<ndl; il++){
		ill=il+nstrtl-1; 
		for(it=0; it<ndt; it++){
		  itt=it+nstrtt-1; 
		  itl=it+il*ndt;
		  zaut[ill] += sttmp->a1[itl]*zu[itt];
		}
	  }
	}
  }
  for(il=ls-1;il<=le-1;il++){
#pragma omp atomic
	zau[il] += zaut[il];
  }
  free(zaut); free(zbut);
}

void c_hacapk_adot_cax_lfmtx_warp_comm
(double *zau, stc_HACApK_lcontrol *st_ctl,
 double *wws, double *wwr, int *isct, int *irct, int nd, double *time_mpi) {
  int ione = 1;
  double one = 1.0;

  double tic;
  int *lpmd = (int*)((size_t)((void*)st_ctl->param) + st_ctl->lpmd_offset); 
  int mpinr = lpmd[2]; 
  int nrank = lpmd[1]; 
  int i;
   
  if (nrank > 1) {
	int *lsp = (int*)((size_t)((void*)st_ctl->param) + st_ctl->lsp_offset);
	int *lnp = (int*)((size_t)((void*)st_ctl->param) + st_ctl->lnp_offset);
	MPI_Comm icomm = MPI_COMM_WORLD;

	int ic;
	int ncdp = (mpinr+1)%nrank;       // my destination neighbor
	int ncsp = (mpinr+nrank-1)%nrank; // my source neighbor
	isct[0] = lnp[mpinr];
	isct[1] = lsp[mpinr];

	// copy local vector to send buffer
	//dlacpy_( "F", &lnp[mpinr], &ione, &zau[lsp[mpinr]-1], &lnp[mpinr], wws, &lnp[mpinr] );
	for(i=0;i<lnp[mpinr];i++)wws[i]=zau[lsp[mpinr]-1+i];
	for (ic=1; ic<nrank; ic++) {
	  MPI_Status stat;
	  tic = MPI_Wtime();
	  // read offset/size from structure
	  int nctp = (ncsp-ic+nrank+1)%nrank; // where it came from
	  irct[0] = lnp[nctp];
	  irct[1] = lsp[nctp];

	  MPI_Status stats[2];
	  MPI_Request reqs[2];
	  if (MPI_SUCCESS != MPI_Isend(wws, isct[0], MPI_DOUBLE, ncdp, nrank+ic, MPI_COMM_WORLD, &reqs[0])) 
		printf( "MPI_Isend failed\n" );
	  if (MPI_SUCCESS != MPI_Irecv(wwr, irct[0], MPI_DOUBLE, ncsp, nrank+ic, MPI_COMM_WORLD, &reqs[1]))
		printf( "MPI_Irecv failed\n" );
	  if (MPI_SUCCESS != MPI_Waitall(2, reqs, stats))
		printf( "MPI_Waitall failed\n" );

	  *time_mpi += (MPI_Wtime()-tic);
	  //blasf77_daxpy( &irct[0], &one, wwr, &ione, &zau[irct[1]-1], &ione );
	  for(i=0;i<irct[0];i++)zau[irct[1]-1+i]+=wwr[i];

	  //dlacpy_( "F", &irct[0], &ione, wwr, &irct[0], wws, &irct[0] );
	  for(i=0;i<irct[0];i++)wws[i]=wwr[i];
	  isct[0] = irct[0];
	  isct[1] = irct[1];
	}
  }
}

extern "C"
void c_hacapk_bicgstab_cax_lfmtx_warp_
(stc_HACApK_leafmtxp *st_leafmtxp, stc_HACApK_lcontrol *st_ctl,
 double *u, double *b, double*param, int *nd, int *nstp, int *lrtrn) {
  // local constants
  int ione = 1;
  double zero =  0.0;
  double one  =  1.0;
  double mone = -1.0;
  // local arrays
  double *zr, *zshdw, *zp, *zt, *zkp, *zakp, *zkt, *zakt;
  double *wws, *wwr;
  int *lpmd = (int*)((size_t)((void*)st_ctl->param) + st_ctl->lpmd_offset);
  int isct[2], irct[2];
  // local variables
  double eps, alpha, beta, zeta, zz, zden, znorm, znormold, bnorm, zrnorm;
  double en_measure_time, st_measure_time, time;
  int info, step, mstep;
  int mpinr, nrank, ierr;
  double time_spmv, time_mpi, time_batch, time_set, time_copy, tic;
  int i, tid;
  MPI_Comm icomm = MPI_COMM_WORLD; //lpmd[0];
  mstep = param[82];
  eps = param[90];
  mpinr = lpmd[2];
  nrank = lpmd[1];
  MPI_Barrier( icomm );

  wws = (double*)malloc((*nd) * sizeof(double));
  wwr = (double*)malloc((*nd) * sizeof(double));

  zt = (double*)malloc((*nd) * sizeof(double));
  zr = (double*)malloc((*nd) * sizeof(double));
  zp = (double*)malloc((*nd) * sizeof(double));
  zkp = (double*)malloc((*nd) * sizeof(double));
  zakp = (double*)malloc((*nd) * sizeof(double));
  zkt = (double*)malloc((*nd) * sizeof(double));
  zakt= (double*)malloc((*nd) * sizeof(double));
  zshdw = (double*)malloc((*nd) * sizeof(double));
  // copy matrix to GPU
  //c_hacapk_adot_body_lfcpy_batch_sorted_(nd, st_leafmtxp);

  time_spmv = 0.0;
  time_mpi = 0.0;
  time_batch = 0.0;
  time_set = 0.0;
  time_copy = 0.0;
  MPI_Barrier( icomm );
  st_measure_time = MPI_Wtime();
  // init
  alpha = 0.0; beta = 0.0; zeta = 0.0;
  zz = 0.0;
#pragma omp parallel for reduction(+:zz)
  for(i=0;i<(*nd);i++){zz += b[i]*b[i];}
  bnorm=sqrt(zz);
#pragma omp parallel for
  for(i=0;i<(*nd);i++)zr[i]=b[i];
  //  .. MATVEC ..
  tic = MPI_Wtime();
  for(i=0;i<(*nd);i++)zshdw[i]=0.0;
  c_hacapk_adot_body_lfmtx_cuda_calc(zshdw,st_leafmtxp,u,wws, &time_batch,&time_set,&time_copy,*nd);
  time_spmv += (MPI_Wtime()-tic);
  c_hacapk_adot_cax_lfmtx_cuda_comm(zshdw, st_ctl, wws, wwr, isct, irct, *nd, &time_mpi);
  //
#pragma omp parallel for
  for(i=0;i<(*nd);i++){
	zr[i]+=mone*zshdw[i];
	zshdw[i]=zr[i];
  }
  zrnorm = 0.0;
#pragma omp parallel for reduction(+:zrnorm)
  for(i=0;i<(*nd);i++)zrnorm += zr[i]*zr[i];
  zrnorm = sqrt(zrnorm);
  if (mpinr == 0) {
	printf( "\n ** BICG (c version, WARP) **\n" );
	printf( "\nOriginal relative residual norm = %.2e/%.2e = %.2e\n",zrnorm,bnorm,zrnorm/bnorm );
	printf( "c_HACApK_bicgstab_cax_lfmtx_warp start\n" );
  }
  for ( step=1; step<=mstep; step++ ) {
	if (zrnorm/bnorm < eps) break;
	// zp(:nd) = zr(:nd) + beta*(zp(:nd) - zeta*zakp(:nd))
	if (beta == zero) {
#pragma omp parallel for
	  for(i=0;i<(*nd);i++)zp[i]=zr[i];
	} else {
#pragma omp parallel for
	  for(i=0;i<(*nd);i++){
	    zp[i] = zr[i] + beta * (zp[i] + zeta*zakp[i]);
	  }
	}
	// zkp(:nd) = zp(:nd)
#pragma omp parallel for
	for(i=0;i<(*nd);i++)zkp[i]=zp[i];
	//  .. MATVEC ..
	//for(i=0;i<(*nd);i++)zakp[i]=0.0;
	tic = MPI_Wtime();
	c_hacapk_adot_body_lfmtx_cuda_calc(zakp,st_leafmtxp,zkp,wws, &time_batch,&time_set,&time_copy,*nd);
	time_spmv += (MPI_Wtime()-tic);
	c_hacapk_adot_cax_lfmtx_cuda_comm(zakp,st_ctl,wws,wwr,isct,irct,*nd, &time_mpi);
	//
	znorm = 0.0;
#pragma omp parallel for reduction(+:znorm)
	for(i=0;i<(*nd);i++)znorm += zshdw[i]*zr[i];
	zden = 0.0;
#pragma omp parallel for reduction(+:zden)
	for(i=0;i<(*nd);i++)zden += zshdw[i]*zakp[i];
	alpha = -znorm/zden;
	znormold = znorm;
	// zt(:nd) = zr(:nd) - alpha*zakp(:nd)
#pragma omp parallel for
	for(i=0;i<(*nd);i++)zt[i]=zr[i]+alpha*zakp[i];
	alpha = -alpha;
	// zkt(:nd) = zt(:nd)
#pragma omp parallel for
	for(i=0;i<(*nd);i++)zkt[i]=zt[i];
	//  .. MATVEC ..
	//for(i=0;i<(*nd);i++)zakt[i]=0.0;
	tic = MPI_Wtime();
	c_hacapk_adot_body_lfmtx_cuda_calc(zakt,st_leafmtxp,zkt,wws, &time_batch,&time_set,&time_copy,*nd);
	time_spmv += (MPI_Wtime()-tic);
	c_hacapk_adot_cax_lfmtx_cuda_comm(zakt,st_ctl,wws,wwr,isct,irct,*nd, &time_mpi);
	//
	znorm = 0.0;
#pragma omp parallel for reduction(+:znorm)
	for(i=0;i<(*nd);i++)znorm += zakt[i]*zt[i];
	zden = 0.0;
#pragma omp parallel for reduction(+:zden)
	for(i=0;i<(*nd);i++)zden += zakt[i]*zakt[i];
	zeta = znorm/zden;
	// u(:nd) = u(:nd) + alpha*zkp(:nd) + zeta*zkt(:nd)
#pragma omp parallel for
	for(i=0;i<(*nd);i++){
	  u[i] += alpha*zkp[i] + zeta*zkt[i];
	}
	// zr(:nd) = zt(:nd) - zeta*zakt(:nd)
	zeta = -zeta;
#pragma omp parallel
	for(i=0;i<(*nd);i++){
	  zr[i]=zt[i] + zeta*zakt[i];
	}
	// beta = alpha/zeta * HACApK_dotp_d(nd,zshdw,zr)/znormold;
	beta = 0.0;
#pragma omp parallel for reduction(+:beta)
	for(i=0;i<(*nd);i++)beta += zshdw[i]*zr[i];
	beta = -alpha/zeta * beta/znormold;
	zrnorm = 0.0;
#pragma omp parallel for reduction(+:zrnorm)
	for(i=0;i<(*nd);i++)zrnorm += zr[i]*zr[i];
        zrnorm = sqrt(zrnorm);
        *nstp = step;
        en_measure_time = MPI_Wtime();
        time = en_measure_time - st_measure_time;
        if (st_ctl->param[0] > 0 && mpinr == 0) {
            printf( " %d: time=%.2e log10(zrnorm/bnorm)=log10(%.2e/%.2e)=%.2e\n",step,time,zrnorm,bnorm,log10(zrnorm/bnorm) );
        }
    }
    MPI_Barrier( icomm );
    en_measure_time = MPI_Wtime();
    time = en_measure_time - st_measure_time;
    if (st_ctl->param[0] > 0) {
        //printf( " End: %d, %.2e\n",mpinr,time );
	  for(i=0;i<nrank;i++){
		if(i==mpinr){
			printf( "C-WARP  %d  BiCG        = %.5e\n", i, time );
			printf( "C-WARP  %d  time_mpi   = %.5e\n", i, time_mpi );
			printf( "C-WARP  %d  time_matvec  = %.5e\n", i, time_spmv );
			printf( "C-WARP  %d  >time_copy  = %.5e\n", i, time_copy );
			printf( "C-WARP  %d  >time_set   = %.5e\n", i, time_set );
			printf( "C-WARP  %d  >time_batch = %.5e\n", i, time_batch );
		}
		MPI_Barrier( icomm );
	  }
    }
    // delete matrix
    //c_hacapk_adot_body_lfdel_batch_(st_leafmtxp);

    // free cpu memory
    free(wws);
    free(wwr);

    free(zt);
    free(zr);
    free(zp);
    free(zkp);
    free(zakp);
    free(zkt);
    free(zakt);
    free(zshdw);
}
